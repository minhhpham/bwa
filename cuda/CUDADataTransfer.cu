#include "kbtree_CUDA.cuh"
#include "CUDADataTransfer.cuh"
#include <stdio.h>
#include <stdint.h>
#include "CUDAKernel_memmgnt.cuh"

char *seq_name_ptr = 0; int seq_name_offset = 0;
char *seq_comment_ptr = 0; int seq_comment_offset = 0;
char *seq_ptr = 0; int seq_offset = 0;
char *seq_qual_ptr = 0; int seq_qual_offset = 0;
char *seq_sam_ptr = 0;
__device__ char *d_seq_sam_ptr = 0;
char *d_seq_name_ptr = 0;
char *d_seq_comment_ptr = 0;
char *d_seq_ptr = 0;
char *d_seq_qual_ptr = 0;
bseq1_t *preallocated_seqs=0, *d_preallocated_seqs=0;

/* Allocate big chunks of strings for seqs and seqs members name, comment, seq, qual */
void CUDAInitSeqsMemory()
{
	// allocate big chunks of memory as pinned memory on host
	gpuErrchk(hipHostMalloc((void**)&seq_name_ptr, SEQ_NAME_LIMIT));
	gpuErrchk(hipHostMalloc((void**)&seq_comment_ptr, SEQ_COMMENT_LIMIT));
	gpuErrchk(hipHostMalloc((void**)&seq_ptr, SEQ_LIMIT));
	gpuErrchk(hipHostMalloc((void**)&seq_qual_ptr, SEQ_QUAL_LIMIT));
	gpuErrchk(hipHostMalloc((void**)&preallocated_seqs, SEQ_MAX_COUNT*sizeof(bseq1_t)));
	gpuErrchk(hipHostMalloc((void**)&seq_sam_ptr, SEQ_SAM_LIMIT));
	// allocate corresponding chunks on device
	gpuErrchk(hipMalloc((void**)&d_seq_name_ptr, SEQ_NAME_LIMIT));
	gpuErrchk(hipMalloc((void**)&d_seq_comment_ptr, SEQ_COMMENT_LIMIT));
	gpuErrchk(hipMalloc((void**)&d_seq_ptr, SEQ_LIMIT));
	gpuErrchk(hipMalloc((void**)&d_seq_qual_ptr, SEQ_QUAL_LIMIT));
	gpuErrchk(hipMalloc((void**)&d_preallocated_seqs, SEQ_MAX_COUNT*sizeof(bseq1_t)));
	char* symbol_addr;
	gpuErrchk(hipGetSymbolAddress((void**)&symbol_addr, d_seq_sam_ptr));
	char* d_temp;
	gpuErrchk(hipMalloc((void**)&d_temp, SEQ_SAM_LIMIT));
	gpuErrchk(hipMemcpy(symbol_addr, &d_temp, sizeof(char*), hipMemcpyHostToDevice));
	fprintf(stderr, "[M::%s] seq name ......... %d MB\n", __func__, (int)SEQ_NAME_LIMIT/1000000);
	fprintf(stderr, "[M::%s] seq comment ...... %d MB\n", __func__, (int)SEQ_COMMENT_LIMIT/1000000);
	fprintf(stderr, "[M::%s] seq  ............. %d MB\n", __func__, (int)SEQ_LIMIT/1000000);
	fprintf(stderr, "[M::%s] seq qual ......... %d MB\n", __func__, (int)SEQ_QUAL_LIMIT/1000000);
	fprintf(stderr, "[M::%s] seq info ......... %d MB\n", __func__, (int)SEQ_MAX_COUNT*sizeof(bseq1_t)/1000000);
	fprintf(stderr, "[M::%s] sam .............. %d MB\n", __func__, (int)SEQ_SAM_LIMIT/1000000);
}

/* transfer one-time static data */
void CUDATransferStaticData(
	const mem_opt_t *opt, 
	const bwt_t *bwt, 
	const bntseq_t *bns, 
	const uint8_t *pac,
	mem_pestat_t *pes0,
	gpu_ptrs_t *gpu_data)
{
		/* CUDA GLOBAL MEMORY ALLOCATION AND TRANSFER */
	fprintf(stderr, "[M::%s] Device memory allocation ......\n", __func__);

	// matching and mapping options (opt)
	fprintf(stderr, "[M::%s] options ...... %.2f MB\n", __func__, (float)sizeof(mem_opt_t)/1000000);
	mem_opt_t* d_opt;
	hipMalloc((void**)&d_opt, sizeof(mem_opt_t));
	hipMemcpy(d_opt, opt, sizeof(mem_opt_t), hipMemcpyHostToDevice);

	// Burrows-Wheeler Transform
		// 1. bwt_t structure
	fprintf(stderr, "[M::%s] bwt .......... %.2f MB\n", __func__, (float)sizeof(bwt_t)/1000000);
	bwt_t* d_bwt;
	hipMalloc((void**)&d_bwt, sizeof(bwt_t));
	hipMemcpy(d_bwt, bwt, sizeof(bwt_t), hipMemcpyHostToDevice);
		// 2. int array of bwt
	fprintf(stderr, "[M::%s] bwt_int ...... %.2f MB\n", __func__, (float)bwt->bwt_size*sizeof(uint32_t)/1000000);
	uint32_t* d_bwt_int ;
	hipMalloc((void**)&d_bwt_int, bwt->bwt_size*sizeof(uint32_t));
	hipMemcpy(d_bwt_int, bwt->bwt, bwt->bwt_size*sizeof(uint32_t), hipMemcpyHostToDevice);
		// 3. int array of Suffix Array
	fprintf(stderr, "[M::%s] suffix array . %.2f MB \n", __func__, (float)bwt->n_sa*sizeof(bwtint_t)/1000000);
	bwtint_t* d_bwt_sa ;
	hipMalloc((void**)&d_bwt_sa, bwt->n_sa*sizeof(bwtint_t));
	hipMemcpy(d_bwt_sa, bwt->sa, bwt->n_sa*sizeof(bwtint_t), hipMemcpyHostToDevice);
		// set pointers on device's memory to bwt_int and SA
	hipMemcpy((void**)&(d_bwt->bwt), &d_bwt_int, sizeof(uint32_t*), hipMemcpyHostToDevice);
	hipMemcpy((void**)&(d_bwt->sa), &d_bwt_sa, sizeof(bwtint_t*), hipMemcpyHostToDevice);

	// BNS
	// First create h_bns as a copy of bns on host
	// Then allocate its member pointers on device and copy data over
	// Then copy h_bns to d_bns
	uint32_t i, size;			// loop index and length of strings
	bntseq_t* h_bns;			// host copy to modify pointers
	h_bns = (bntseq_t*)malloc(sizeof(bntseq_t));
	memcpy(h_bns, bns, sizeof(bntseq_t));
	h_bns->anns = (bntann1_t*)malloc(bns->n_seqs*sizeof(bntann1_t));
	memcpy(h_bns->ambs, bns->ambs, bns->n_holes*sizeof(bntamb1_t));
	h_bns->ambs = (bntamb1_t*)malloc(bns->n_holes*sizeof(bntamb1_t));
	memcpy(h_bns->anns, bns->anns, bns->n_seqs*sizeof(bntann1_t));

		// allocate anns.name
	for (i=0; i<bns->n_seqs; i++){
		size = strlen(bns->anns[i].name);
		// allocate this name and copy to device
		hipMalloc((void**)&(h_bns->anns[i].name), size+1); 			// +1 for "\0"
		hipMemcpy(h_bns->anns[i].name, bns->anns[i].name, size+1, hipMemcpyHostToDevice);
	}
	// allocate anns.anno
	for (i=0; i<bns->n_seqs; i++){
		size = strlen(bns->anns[i].anno);
		// allocate this name and copy to device
		hipMalloc((void**)&(h_bns->anns[i].anno), size+1); 			// +1 for "\0"
		hipMemcpy(h_bns->anns[i].anno, bns->anns[i].anno, size+1, hipMemcpyHostToDevice);
	}
		// now h_bns->anns has pointers of name and anno on device
		// allocate anns on device and copy data from h_bns->anns to device
	bntann1_t* temp_d_anns;
	fprintf(stderr, "[M::%s] bns.anns ..... %.2f MB\n", __func__, (float)bns->n_seqs*sizeof(bntann1_t)/1000000);
	hipMalloc((void**)&temp_d_anns, bns->n_seqs*sizeof(bntann1_t));
	hipMemcpy(temp_d_anns, h_bns->anns, bns->n_seqs*sizeof(bntann1_t), hipMemcpyHostToDevice);
		// now assign this pointer to h_bns->anns
	h_bns->anns = temp_d_anns;

		// allocate bns->ambs on device and copy data to device
	fprintf(stderr, "[M::%s] bns.ambs ..... %.2f MB\n", __func__, (float)bns->n_holes*sizeof(bntamb1_t)/1000000);
	hipMalloc((void**)&h_bns->ambs, bns->n_holes*sizeof(bntamb1_t));
	hipMemcpy(h_bns->ambs, bns->ambs, bns->n_holes*sizeof(bntamb1_t), hipMemcpyHostToDevice);

		// finally allocate d_bns and copy from h_bns
	fprintf(stderr, "[M::%s] bns .......... %.2f MB\n", __func__, (float)sizeof(bntseq_t)/1000000);
	bntseq_t* d_bns;
	hipMalloc((void**)&d_bns, sizeof(bntseq_t));
	hipMemcpy(d_bns, h_bns, sizeof(bntseq_t), hipMemcpyHostToDevice);

	// PAC
	fprintf(stderr, "[M::%s] pac .......... %.2f MB\n", __func__, (float)bns->l_pac*sizeof(uint8_t)/1000000);
	uint8_t* d_pac ;
	hipMalloc((void**)&d_pac, bns->l_pac/4*sizeof(uint8_t)); 		// l_pac is length of ref seq
	hipMemcpy(d_pac, pac, bns->l_pac/4*sizeof(uint8_t), hipMemcpyHostToDevice); 		// divide by 4 because 2-bit encoding

	// paired-end stats: only allocate on device
	mem_pestat_t* d_pes;
	if (opt->flag&MEM_F_PE){
		fprintf(stderr, "[M::%s] pestat ....... %.2f MB\n", __func__, (float)4*sizeof(mem_pestat_t)/1000000);
		hipMalloc((void**)&d_pes, 4*sizeof(mem_pestat_t));
	}

	// output
	gpu_data->d_opt = d_opt;
	gpu_data->d_bwt = d_bwt;
	gpu_data->d_bns = d_bns;
	gpu_data->d_pac = d_pac;
	gpu_data->d_pes = d_pes;
	gpu_data->h_pes0 = pes0;
}

/* transfer seqs */
void CUDATransferSeqs(int n_seqs)
{
	// copy name to device
	gpuErrchk(hipMemcpy(d_seq_name_ptr, seq_name_ptr, seq_name_offset, hipMemcpyHostToDevice));
	// copy seq to device
	gpuErrchk(hipMemcpy(d_seq_ptr, seq_ptr, seq_offset, hipMemcpyHostToDevice));
	// copy comment to device
	gpuErrchk(hipMemcpy(d_seq_comment_ptr, seq_comment_ptr, seq_comment_offset, hipMemcpyHostToDevice));
	// copy qual to device
	gpuErrchk(hipMemcpy(d_seq_qual_ptr, seq_qual_ptr, seq_qual_offset, hipMemcpyHostToDevice));
	// copy seqs to device
	gpuErrchk(hipMemcpy(d_preallocated_seqs, preallocated_seqs, n_seqs*sizeof(bseq1_t), hipMemcpyHostToDevice));
}

/* transfer SAM output */

void CUDADataFree(){
	hipHostFree(preallocated_seqs);
	hipFree(d_preallocated_seqs);
	hipHostFree(seq_name_ptr); hipHostFree(seq_comment_ptr), hipHostFree(seq_ptr), hipHostFree(seq_qual_ptr); hipHostFree(seq_sam_ptr);
	hipFree(d_seq_name_ptr); hipFree(d_seq_comment_ptr); hipFree(d_seq_ptr); hipFree(d_seq_qual_ptr);
}